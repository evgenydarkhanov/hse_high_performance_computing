#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int N = 1024;

void fill_matrix(double *matrix, int rows, int cols)
{
    for (int i = 0; i < cols * rows; i++)
    {
        matrix[i] = rand() % 100;
    }
}

void matmul_cpu(int rowsA, int colsA, int colsB, double *A, double *B, double *C)
{
    for (int i = 0; i < rowsA; i++)
    {
        for (int j = 0; j < colsB; j++)
        {
            C[i * colsB + j] = 0;
            for (int k = 0; k < colsA; k++)
            {
                C[i * colsB + j] += A[i * colsA + k] * B[k * colsB + j];

            }
        }
    }
}

__global__ void matmul_gpu(double *A, double *B, double *C, int colsA, int colsB)
{
    int idx_a = blockDim.y * blockIdx.y + threadIdx.y;  // row from A
    int idx_b = blockDim.x * blockIdx.x + threadIdx.x;  // col from B
    int idx_c = colsA * idx_a + idx_b;                  // elem from C
    
    if (idx_a < N && idx_b < N)
    {
        double sum = 0;
        for (int k = 0; k < colsA; k++)
        {
            sum += A[idx_a * colsA + k] * B[idx_b + k * colsB];
        }
        C[idx_c] = sum;
    }
}

int main()
{
    int cols = N;
    int rows = N;
    size_t matrix_size = cols * rows * sizeof(double);

    // creating host matrices
    double *h_A, *h_B, *h_C;
    hipHostAlloc( (void**)&h_A, matrix_size, hipHostMallocDefault );
    hipHostAlloc( (void**)&h_B, matrix_size, hipHostMallocDefault );
    hipHostAlloc( (void**)&h_C, matrix_size, hipHostMallocDefault );

    // filling matrices with random numbers
    fill_matrix(h_A, rows, cols); 
    fill_matrix(h_B, rows, cols);

    // creating device matrices
    double *d_A, *d_B, *d_C;
    hipMalloc( (void**)&d_A, matrix_size );
    hipMalloc( (void**)&d_B, matrix_size );
    hipMalloc( (void**)&d_C, matrix_size );
    
    // block and grid sizes
    int BLOCK_SIZE = 32;
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocksPerGrid = dim3(GRID_SIZE, GRID_SIZE, 1);

    // time
    hipEvent_t start_gpu, stop_gpu;
    float gpu_time;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    // copying the data
    hipEventRecord(start_gpu, 0);
    hipMemcpy( d_A, h_A, matrix_size, hipMemcpyHostToDevice );
    hipMemcpy( d_B, h_B, matrix_size, hipMemcpyHostToDevice );
    //cudaMemcpy( d_C, h_C, matrix_size, cudaMemcpyHostToDevice );

    // kernel
    matmul_gpu<<< blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, cols, cols);
    
    hipMemcpy( h_C, d_C, matrix_size, hipMemcpyDeviceToHost );
    hipEventRecord(stop_gpu, 0);
    
    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);

    printf("GPU time (pinned memory, copyHTD, kernel, copyDTH): %.3f ms\n", gpu_time);
    
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    /*
    // checking with CPU multiplication
    double *h_D = (double*)malloc(matrix_size);
    
    cudaEvent_t start_cpu, stop_cpu;
    float cpu_time;
    cudaEventCreate(&start_cpu);
    cudaEventCreate(&stop_cpu);

    cudaEventRecord(start_cpu, 0);
    matmul_cpu(rows, cols, cols, h_A, h_B, h_D);
    cudaEventRecord(stop_cpu, 0);

    cudaEventSynchronize(stop_cpu);
    cudaEventElapsedTime(&cpu_time, start_cpu, stop_cpu);

    printf("CPU time: %.3f ms\n", cpu_time);
    
    cudaEventDestroy(start_cpu);
    cudaEventDestroy(stop_cpu);
    
    printf("checking\n");
    double delta = 0;
    for (int i = 0; i < cols * rows; i++)
    {
        delta += fabs(h_D[i] - h_C[i]);
    }
    if (delta > 0.00001)
    {
        printf("error %f\n", delta);
    }
    else
    {
        printf("good %f\n", delta);
    }
    free(h_D);
    */
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
