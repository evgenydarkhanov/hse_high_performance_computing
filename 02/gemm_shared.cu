#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int N = 1024;
const int BLOCK_SIZE = 32;
const int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
void fill_matrix(double *matrix, int rows, int cols)
{
    for (int i = 0; i < cols * rows; i++)
    {
        matrix[i] = rand() % 100;
    }
}

void matmul_cpu(int rowsA, int colsA, int colsB, double *A, double *B, double *C)
{
    for (int i = 0; i < rowsA; i++)
    {
        for (int j = 0; j < colsB; j++)
        {
            C[i * colsB + j] = 0;
            for (int k = 0; k < colsA; k++)
            {
                C[i * colsB + j] += A[i * colsA + k] * B[k * colsB + j];

            }
        }
    }
}

__global__ void matmul_gpu(double *A, double *B, double *C, int size) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Разделяемая память для подматриц
    __shared__ double shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double shared_B[BLOCK_SIZE][BLOCK_SIZE];
    
    double sum = 0.0;

    // Перемножение блоков
    for (int k = 0; k < (size + BLOCK_SIZE - 1) / BLOCK_SIZE; k++) 
    {
        // Загрузка данных в разделяемую память
        if (row < size && (k * BLOCK_SIZE + threadIdx.x) < size) 
        {
            shared_A[threadIdx.y][threadIdx.x] = A[row * size + (k * BLOCK_SIZE + threadIdx.x)];
        }
        else
        {
            shared_A[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (col < size && (k * BLOCK_SIZE + threadIdx.y) < size)
        {
            shared_B[threadIdx.y][threadIdx.x] = B[(k * BLOCK_SIZE + threadIdx.y) * size + col];
        }
        else
        {
            shared_B[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads(); // Синхронизация потоков
        
        // Умножение блоков
        for (int n = 0; n < BLOCK_SIZE; n++)
        {
            sum += shared_A[threadIdx.y][n] * shared_B[n][threadIdx.x];
        }

        __syncthreads(); // Синхронизация потоков
     }

     // Запись результата в глобальную память
     if (row < size && col < size)
     {
        C[row * size + col] = sum;
     }
}

int main()
{
    int cols = N;
    int rows = N;
    size_t matrix_size = cols * rows * sizeof(double);

    // creating host matrices
    double *h_A = (double*)malloc(matrix_size);
    double *h_B = (double*)malloc(matrix_size);
    double *h_C = (double*)malloc(matrix_size);

    // filling matrices with random numbers
    fill_matrix(h_A, rows, cols); 
    fill_matrix(h_B, rows, cols);

    // creating device matrices
    double *d_A, *d_B, *d_C;
    hipMalloc( (void**)&d_A, matrix_size );
    hipMalloc( (void**)&d_B, matrix_size );
    hipMalloc( (void**)&d_C, matrix_size );
    
    // block and grid sizes
    dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocksPerGrid = dim3(GRID_SIZE, GRID_SIZE, 1);

    // time
    hipEvent_t start_gpu, stop_gpu;
    float gpu_time;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    // copying the data
    hipEventRecord(start_gpu, 0);
    hipMemcpy( d_A, h_A, matrix_size, hipMemcpyHostToDevice );
    hipMemcpy( d_B, h_B, matrix_size, hipMemcpyHostToDevice );
    // cudaMemcpy( d_C, h_C, matrix_size, cudaMemcpyHostToDevice );

    // kernel
    matmul_gpu<<< blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, cols);

    hipMemcpy( h_C, d_C, matrix_size, hipMemcpyDeviceToHost );
    hipEventRecord(stop_gpu, 0);

    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);

    printf("GPU time (shared memory, copyHTD, kernel, copyDTH): %.3f ms\n", gpu_time);

    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    /*
    // checking with CPU multiplication
    double *h_D = (double*)malloc(matrix_size);
    
    cudaEvent_t start_cpu, stop_cpu;
    float cpu_time;
    cudaEventCreate(&start_cpu);
    cudaEventCreate(&stop_cpu);

    cudaEventRecord(start_cpu, 0);
    matmul_cpu(rows, cols, cols, h_A, h_B, h_D);
    cudaEventRecord(stop_cpu, 0);

    cudaEventSynchronize(stop_cpu);
    cudaEventElapsedTime(&cpu_time, start_cpu, stop_cpu);

    printf("CPU time: %.3f ms\n", cpu_time);
    
    cudaEventDestroy(start_cpu);
    cudaEventDestroy(stop_cpu);

    printf("checking\n");
    double delta = 0;
    for (int i = 0; i < cols * rows; i++)
    {
        delta += fabs(h_D[i] - h_C[i]);
    }
    if (delta > 0.00001)
    {
        printf("error %f\n", delta);
    }
    else
    {
        printf("good %f\n", delta);
    }
    free(h_D);
    */
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
