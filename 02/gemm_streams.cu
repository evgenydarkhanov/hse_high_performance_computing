#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const int N = 1024;

void fill_matrix(double *matrix, int rows, int cols)
{
    for (int i = 0; i < cols * rows; i++)
    {
        matrix[i] = rand() % 10 + 1;
    }
}

void matmul_cpu(int rowsA, int colsA, int colsB, double *A, double *B, double *C)
{
    for (int i = 0; i < rowsA; i++)
    {
        for (int j = 0; j < colsB; j++)
        {
            C[i * colsB + j] = 0;
            for (int k = 0; k < colsA; k++)
            {
                C[i * colsB + j] += A[i * colsA + k] * B[k * colsB + j];

            }
        }
    }
}

__global__ void matmul_gpu(double *A, double *B, double *C, int size)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;  // row from A
    int col = blockDim.x * blockIdx.x + threadIdx.x;  // col from B
    
    if (row < size && col < size)
    {
        double sum = 0;
        for (int k = 0; k < size; k++)
        {
            sum += A[row * size + k] * B[k * size + col];
        }
        C[row * size + col] = sum;
    }
}

int main()
{
    int cols = N;
    int rows = N;
    size_t matrix_size = cols * rows * sizeof(double);

    // creating host matrices
    double *h_A = (double*)malloc(matrix_size);
    double *h_B = (double*)malloc(matrix_size);
    double *h_C = (double*)malloc(matrix_size);

    // filling matrices with random numbers
    fill_matrix(h_A, rows, cols); 
    fill_matrix(h_B, rows, cols);

    // block and grid sizes
    int BLOCK_SIZE = 32;
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocksPerGrid( (N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

    // creating streams
    int num_streams = 4;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++)
    {
        hipStreamCreate(&streams[i]);
    }

    // timing
    hipEvent_t start_gpu, stop_gpu;
    float gpu_time;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start_gpu, 0);

    // creating device matrices
    double *d_A[num_streams], *d_B, *d_C[num_streams];
    for (int i = 0; i < num_streams; i++)
    {
        hipMalloc( (void**)&d_A[i], matrix_size/num_streams );
        hipMalloc( (void**)&d_C[i], matrix_size/num_streams );

        hipMemcpyAsync( d_A[i], h_A + i * (N/num_streams)*N, matrix_size/num_streams, hipMemcpyHostToDevice, streams[i] );
    }

    hipMalloc( (void**)&d_B, matrix_size/num_streams );
    hipMemcpyAsync( d_B, h_B, matrix_size, hipMemcpyHostToDevice );
    
    // kernel
    for (int i = 0; i < num_streams; i++)
    {
        matmul_gpu<<< blocksPerGrid, threadsPerBlock, 0, streams[i] >>>(d_A[i], d_B, d_C[i], N/num_streams);
        hipMemcpyAsync( h_C + i * (N/num_streams)*N, d_C[i], matrix_size/num_streams, hipMemcpyDeviceToHost, streams[i] );
        //cudaStreamSynchronize(streams[i]);
    }

    for (int i = 0; i < num_streams; i++)
    {
        hipStreamSynchronize(streams[i]);
        hipFree(d_A[i]);
        hipFree(d_C[i]);
        hipStreamDestroy(streams[i]);
    }

    hipFree(d_B);
    hipEventRecord(stop_gpu, 0);

    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);

    printf("GPU time (streams, copyHTD, kernel, copyDTH): %.3f ms\n", gpu_time);

    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);

    /*
    // checking with CPU multiplication
    double *h_D = (double*)malloc(matrix_size);
    
    cudaEvent_t start_cpu, stop_cpu;
    float cpu_time;
    cudaEventCreate(&start_cpu);
    cudaEventCreate(&stop_cpu);

    cudaEventRecord(start_cpu, 0);
    matmul_cpu(rows, cols, cols, h_A, h_B, h_D);
    cudaEventRecord(stop_cpu, 0);

    cudaEventSynchronize(stop_cpu);
    cudaEventElapsedTime(&cpu_time, start_cpu, stop_cpu);

    printf("CPU time: %.3f ms\n", cpu_time);
    
    cudaEventDestroy(start_cpu);
    cudaEventDestroy(stop_cpu);

    printf("checking\n");

    double delta = 0;
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            delta += fabs(h_D[i * cols + j] - h_C[i * cols + j]);
        }
    }
    if (delta > 0.00001)
    {
        printf("error %f\n", delta);
    }
    else
    {
        printf("good %f\n", delta);
    }
    free(h_D);
    */
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
